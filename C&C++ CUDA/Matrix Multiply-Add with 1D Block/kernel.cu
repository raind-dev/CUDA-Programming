#include "hip/hip_runtime.h"
﻿
#include "hip/hip_runtime.h"
#include ""

#include <stdio.h>
#include <iostream>

__global__ void traverse_device_data(int *device) {
	int x = threadIdx.x + blockIdx.x * blockDim.x;
	int y = threadIdx.y + blockIdx.y * blockDim.y;
	int idx = x + y * blockDim.x;
    printf("%d ", device[idx]);
}

__global__ void matrix_operations(int* mat, int* mul, int* scl, int* res, int N, int K, int W) {
    int x = threadIdx.x + blockIdx.x * blockDim.x;
    for (int i = 0; i < K; i++) {
        int tmp = 0;
        for (int j = 0; j < N; j++) {
            tmp += mat[x * N + j] * mul[i + j * K];
        }
        tmp += scl[x * W + i];
        res[x * W + i] = tmp;
    }
}

bool is_cuda_available()
{
    int device = 0;
    hipError_t status = hipSetDevice(device); // hipGetDeviceCount
    if (status != hipSuccess) {
        fprintf(stderr, "Setting device failed! Please check whether CUDA-Capable GPU installed.\n");
        return false;
    }
    return true;
}

template<size_t C, size_t R>
bool allocate_host_data(int (&map)[C][R], const char* name)
{
    srand(time(NULL));
    printf("%s: \n", name);
    for (int i = 0; i < C; i++) {
        for (int j = 0; j < R; j++) {
            map[i][j] = rand() % 2;
            printf("%d ", map[i][j]);
        }
        printf("\n");
    }
    printf("\n");
    return true;
}

bool allocate_device_data(int** device_buffer, int col, int row)
{
	hipError_t status = hipMalloc(device_buffer, sizeof(int) * col * row);
	if (status != hipSuccess) {
		fprintf(stderr, "Allocate device data failed: %d\n", status);
		return false;
	}
    return true;
}

template<size_t C, size_t R>
bool copy_host_data_to_device(int (&host_buffer)[C][R], int** device_buffer, const char* name)
{
	hipError_t status = hipMemcpy(*device_buffer, host_buffer, sizeof(int) * C * R, hipMemcpyHostToDevice);
    if (status != hipSuccess) {
		fprintf(stderr, "Copy host data %s to device failed: %d\n", name, status);
		return false;
    }
    return true;
}

int main()
{
    if (!is_cuda_available()) {
        return -1;
    }
    const int col = 3, row = 3;
    int mat[col][row];
    int mul[row][col];
    int scl[col][col];
    int res[9] = { 0 };
    int* gpu_mat = nullptr;
    int* gpu_mul = nullptr;
    int* gpu_scl = nullptr;
    int* gpu_res = nullptr;
    if (!allocate_host_data(mat, "Matrix") ||
        !allocate_host_data(mul, "Multiplier") ||
        !allocate_host_data(scl, "Scalar")) {
        return -1;
    }
    if (!allocate_device_data(&gpu_mat, col, row) || 
        !allocate_device_data(&gpu_mul, row, col) ||
        !allocate_device_data(&gpu_scl, col, col) ||
        !allocate_device_data(&gpu_res, col, col)) {
        return -1;
    }
	if (!copy_host_data_to_device(mat, &gpu_mat, "Matrix") ||
        !copy_host_data_to_device(mul, &gpu_mul, "Multiplier") ||
        !copy_host_data_to_device(scl, &gpu_scl, "Scalar")) {
        return -1;
    }
    dim3 grid = 1;
    dim3 block(3, 3);
    printf("Traverse GPU Matrix:\n");
	traverse_device_data << <grid, block >> > (gpu_mat);
    block = 3;
    matrix_operations << < grid, block >> > (gpu_mat, gpu_mul, gpu_scl, gpu_res, row, col, col);
    hipDeviceSynchronize();

    hipError_t status = hipMemcpy(res, gpu_res, sizeof(int) * col * col, hipMemcpyDeviceToHost);
    if (status != hipSuccess) {
        fprintf(stderr, "Can not copy the results from device to host\n");
        return -1;
    }
    printf("\n\nThe Results: \n");
    for (int i = 0; i < sizeof(res) / sizeof(int); i++) {
        printf("%d ", res[i]);
        if (i % col == 2) {
            printf("\n");
        }
    }

    hipFree(gpu_mat);
	hipFree(gpu_mul);
	hipFree(gpu_scl);
    hipFree(gpu_res);

    return 0;
}
